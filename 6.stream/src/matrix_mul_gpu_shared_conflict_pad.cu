#include "hip/hip_runtime.h"
#include <stdio.h>
#include <utils.hpp>

#define BLOCKSIZE 16

__global__ void matrix_mul_kernel_shared_static_with_conflict_pad(float *M_device, float *N_device, float *P_device, int width,int blk_size) {
    // pad操作，在分配的时候给共享矩阵最后一列后增加了一行空列，让原本conflict的位置错开，但是并不是所有的都错开了，只是得到了一定的缓解
    __shared__ float M_shared[BLOCKSIZE][BLOCKSIZE + 1];   
    __shared__ float N_shared[BLOCKSIZE][BLOCKSIZE + 1];

    // 在之前的方法的基础上，只要将tx和ty的索引交换位置，就产生了bank conflict
    // 通俗理解：一个block中有 blk_size*blk_size 个线程，现在让每个线程负责计算其对应位置的转置位置的运算，只需要交换block内部的局部索引即可
    int ty = threadIdx.x;     
    int tx = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;
    float element_sum = 0.0;     // 定义一个变量来存放计算结果

    if (x < width && y < width) {
        for (int tile_idx = 0; tile_idx < width / blk_size; tile_idx++) {
            // printf("tile_num: %d\n", width / BLOCKSIZE);
            M_shared[ty][tx] = M_device[y * width + (tile_idx * blk_size + tx)];
            N_shared[ty][tx] = N_device[((tile_idx * blk_size + ty)) * width + x];
            __syncthreads();

            for (int k = 0; k < blk_size; k++){
                element_sum += M_shared[ty][k] * N_shared[k][tx];
            }
            __syncthreads();
        }
        P_device[y * width + x] = element_sum;
    }   
}


__global__ void matrix_mul_kernel_shared_dynamic_with_conflict_pad(float *M_device, float *N_device, float *P_device, int width,int blk_size) {
    /* 
        声明动态共享变量的时候需要加extern，同时需要是一维的 
        注意这里有个坑, 不能够像这样定义： 
            __shared__ float M_deviceShared[];
            __shared__ float N_deviceShared[];
        因为在cuda中定义动态共享变量的话，无论定义多少个他们的地址都是一样的。
        所以如果想要像上面这样使用的话，需要用两个指针分别指向shared memory的不同位置才行
    */
   
    //    这个变量实际上是在内核调用时由外部（例如，在主机代码中或在调用内核之前）分配的，而不是在内核函数内部自动分配的。
    extern __shared__ float shared_mem[];    // 注意动态的shared memory矩阵是一维的
    // 等价于在分配的时候给共享矩阵最后一列后增加了一行空列，让原本conflict的位置错开
    int shared_mem_stride = (blk_size + 1) * blk_size;

    int ty = threadIdx.x;     
    int tx = threadIdx.y;
    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;
    
    float element_sum = 0.0;     // 定义一个变量来存放计算结果
    if (x < width && y < width) {
        for (int tile_idx = 0; tile_idx < width / blk_size; tile_idx++) {
            // printf("tile_num: %d\n", width / BLOCKSIZE);
            // 动态的就将静态的两个shared_mem矩阵放在一起，然后展平，同一个索引对应的两个矩阵的元素的位置相差shared_mem_stride
            shared_mem[ty * (blk_size + 1) + tx] = M_device[y * width + (tile_idx * blk_size + tx)];
            shared_mem[(ty * (blk_size + 1) + tx) + shared_mem_stride] = N_device[((tile_idx * blk_size + ty)) * width + x];
            __syncthreads();

            for (int k = 0; k < blk_size; k++){
                element_sum += shared_mem[ty * (blk_size + 1) + k] * shared_mem[(k * (blk_size + 1) + tx) + shared_mem_stride];
            }
            __syncthreads();
        }
        P_device[y * width + x] = element_sum;
    }   
}


void matrix_mul_gpu_shared_with_conflict_pad(float* M_host, float* N_host, float* P_host, int width, int blk_size, bool use_static_shared_memory) {
    float *M_device, *N_device, *P_device;   // device上的输入矩阵和输出矩阵

    // 申请device上的内存
    hipMalloc((void**)&M_device, width * width * sizeof(float));
    hipMalloc((void**)&N_device, width * width * sizeof(float));
    hipMalloc((void**)&P_device, width * width * sizeof(float));

    // 将host上的输入矩阵拷贝到device上
    hipMemcpy(M_device, M_host, width * width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(N_device, N_host, width * width * sizeof(float), hipMemcpyHostToDevice);

    // 调用kernel函数
    dim3 block_size(blk_size, blk_size);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (width + block_size.y - 1) / block_size.y);
    LAST_KERNEL_CHECK();
    if (use_static_shared_memory){
        matrix_mul_kernel_shared_static_with_conflict_pad<<<grid_size, block_size>>>(M_device, N_device, P_device, width, blk_size);
    }
    else{
        matrix_mul_kernel_shared_dynamic_with_conflict_pad<<<grid_size, block_size, (blk_size + 1) * blk_size * sizeof(float)*2, nullptr>>>(M_device, N_device, P_device, width, blk_size);
    }
    

    LAST_KERNEL_CHECK();                  // 检查同步错误，即内核函数matrix_mul_kernel()调用时刻的错误
    CUDACHECK(hipDeviceSynchronize());   // 检查异步错误，即内核函数matrix_mul_kernel()执行过程中的错误
    LAST_KERNEL_CHECK();   


    // 将device上的输出矩阵拷贝到host上
    CUDACHECK(hipMemcpy(P_host, P_device, width * width * sizeof(float), hipMemcpyDeviceToHost));
    LAST_KERNEL_CHECK();

    // 释放device上的内存
    hipFree(M_device);
    hipFree(N_device); 
    hipFree(P_device);
}