// rector_addition.cu

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addVectors(int* a, int* b, int* c, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int size = 32;
    int a[size], b[size], c[size];
    int *d_a, *d_b, *d_c;

    // Allocate memory on device
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Initialize arrays a and b, and copy them to device
    for (int i = 0; i < size; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int blocksize = 1024;    // 设置超过1024时，会报错
    int girdsize = (size + blocksize - 1) / blocksize;
    addVectors<<<girdsize, blocksize>>>(d_a, d_b, d_c, size);

    // Copy result back to host and free device memory
    // Copy result back to host                
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
} 
